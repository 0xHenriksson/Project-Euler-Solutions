/*
The following iterative sequence is defined for the set of positive integers:

n -> n/2 (n is even)
n -> 3n+1 (n is odd)

Using the rule above and starting with 13, we generate the following sequence:
13->40->20->10->5->16->8->4->2->1

It can be seen that this sequence (starting at 13) and finishing at 1) contains 10 terms. Although it has not been proved yet (Collatz Problem), it is thought that all starting numbers finish at 1.

Which starting number, under one million, produces the longest chain?

Once the chain starts the terms are allowed to go above one million.
*/



#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>

__device__ unsigned int collatz_length(unsigned int n) {
    unsigned int length = 1;
    while (n > 1) {
        if (n % 2 == 0) {
            n /= 2;
        } else {
            n = 3 * n + 1;
        }
        length++;
    }
    return length;
}


__global__ void calculate_collatz(unsigned int* start_numbers, unsigned int* lengths, int num_numbers) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_numbers) {
        lengths[start_numbers[i]-1] = collatz_length(start_numbers[i]);
    }
}

int main() {
    const int limit = 1000000;
    std::vector<unsigned int> start_numbers(limit);
    for (unsigned int i = 0; i < limit; ++i) {
        start_numbers[i] = i + 1;
    }

    unsigned int* d_start_numbers;
    unsigned int* d_lengths;

    hipMallocManaged(&d_start_numbers, limit * sizeof(unsigned int));
    hipMallocManaged(&d_lengths, limit * sizeof(unsigned int));

    // Copy start numbers to device
    hipMemcpy(d_start_numbers, start_numbers.data(), limit * sizeof(unsigned int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (limit + threadsPerBlock - 1) / threadsPerBlock;

    calculate_collatz<<<blocksPerGrid, threadsPerBlock>>>(d_start_numbers, d_lengths, limit);

    hipDeviceSynchronize();

    std::vector<unsigned int> lengths(limit);
    hipMemcpy(lengths.data(), d_lengths, limit * sizeof(unsigned int), hipMemcpyDeviceToHost);

    unsigned int max_length = 0;
    unsigned int max_start_number = 0;
    for (unsigned int i = 0; i < limit; ++i) {
        if (lengths[i] > max_length) {
            max_length = lengths[i];
            max_start_number = i + 1;
        }
    }

    std::cout << "The starting number with the longest chain is: " << max_start_number << std::endl;
    std::cout << "The length of the chain is: " << max_length << std::endl;

    hipFree(d_start_numbers);
    hipFree(d_lengths);

    return 0;
}